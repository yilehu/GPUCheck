
#include "hip/hip_runtime.h"


int BlockDim()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.maxThreadsPerBlock;
}
int GridDim()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.maxGridSize[0];
}
int major()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.major;
}
int minor()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.minor;
}